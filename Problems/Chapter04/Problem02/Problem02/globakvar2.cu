#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// function for checking the CUDA runtime API results.
inline
void checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess)
	{
		printf_s("Error: %s : %d", __FILE__, __LINE__);
		printf_s("CUDA Runtime Error: %d: %s\n", result, hipGetErrorString(result));
		exit(1);
	}
#endif
}

__device__ float devData[1];

__global__ void checkGlobalVar()
{
	// display the original value
	printf("Device:  value of global variable is %f\n", devData[0]);

	// alter global value
	devData[0] += 2.0f;
}

int main(void)
{
	// initialize variables
	float  value = 3.14f;
	float  *devPtr = NULL;
	
	// get address and copy to device
	checkCuda(hipGetSymbolAddress((void**)&devPtr, devData));
	checkCuda(hipMemcpy(devPtr, &value, sizeof(float), hipMemcpyHostToDevice));
	printf_s("Host:  copied %f to the global variable\n", value);

	// invoke kernel
	checkGlobalVar <<<1, 1>>> ();
	
	// copy variable back to host
	checkCuda(hipMemcpy(&value, devPtr, sizeof(float), hipMemcpyDeviceToHost));
	printf_s("Host:  the value changed by the kernel to %f\n", value);

	checkCuda(hipDeviceReset());

	return EXIT_SUCCESS;

}