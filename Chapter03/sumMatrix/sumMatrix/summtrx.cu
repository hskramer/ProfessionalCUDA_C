#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


inline
void checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess)
	{
		printf("Error: %s : %d", __FILE__, __LINE__);
		printf("CUDA Runtime Error: %d: %s\n", result, hipGetErrorString(result));
		exit(1);
	}
#endif
}



void initialData(float *ip, const int size)
{
	int i;

	for (i = 0; i < size; i++)
	{
		ip[i] = (float)(rand() & 0xFF) / 10.0f;
	}
}

void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny)
{
	float *ia = A;
	float *ib = B;
	float *ic = C;

	for (int iy = 0; iy < ny; iy++)
	{
		for (int ix = 0; ix < nx; ix++)
		{
			ic[ix] = ia[ix] + ib[ix];
		}

		ia += nx;
		ib += nx;
		ic += nx;
	}

	return;
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
	double epsilon = 1.0E-8;

	for (int i = 0; i < N; i++)
	{
		if (abs(hostRef[i] - gpuRef[i]) > epsilon)
		{
			printf("host %f gpu %f ", hostRef[i], gpuRef[i]);
			printf("Arrays do not match.\n\n");
			break;
		}
	}
}

// grid 2D block 2D
__global__ void sumMatrixOnGPU2D(float *A, float *B, float *C, int NX, int NY)
{
	unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int idx = iy * NX + ix;

	if (ix < NX && iy < NY)
	{
		C[idx] = A[idx] + B[idx];
	}
}

int main(int argc, char **argv)
{
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	checkCuda(hipGetDeviceProperties(&deviceProp, dev));
	checkCuda(hipSetDevice(dev));

	// set up data size of matrix
	int nx = 1 << 14;
	int ny = 1 << 14;

	int nxy = nx * ny;
	int nBytes = nxy * sizeof(float);

	// malloc host memory
	float *h_A, *h_B, *hostRef, *gpuRef;
	h_A = (float *)malloc(nBytes);
	h_B = (float *)malloc(nBytes);
	hostRef = (float *)malloc(nBytes);
	gpuRef = (float *)malloc(nBytes);

	// initialize data at host side
	initialData(h_A, nxy);
	initialData(h_B, nxy);


	// add matrix at host side for result checkCudas
	sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);

	// malloc device global memory
	float *d_MatA, *d_MatB, *d_MatC;
	checkCuda(hipMalloc((void **)&d_MatA, nBytes));
	checkCuda(hipMalloc((void **)&d_MatB, nBytes));
	checkCuda(hipMalloc((void **)&d_MatC, nBytes));

	// transfer data from host to device
	checkCuda(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int dimx = 32;
	int dimy = 32;

	if (argc > 2)
	{
		dimx = atoi(argv[1]);
		dimy = atoi(argv[2]);
	}

	dim3 block(dimx, dimy);
	dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

	// execute the kernel
	checkCuda(hipDeviceSynchronize());
	sumMatrixOnGPU2D << <grid, block >> >(d_MatA, d_MatB, d_MatC, nx, ny);
	checkCuda(hipDeviceSynchronize());

	printf("sumMatrixOnGPU2D <<<(%d,%d), (%d,%d)>>>\n", grid.x, grid.y, block.x, block.y);
	checkCuda(hipGetLastError());

	// copy kernel result back to host side
	checkCuda(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost));

	// check device results
	checkResult(hostRef, gpuRef, nxy);

	// free device global memory
	checkCuda(hipFree(d_MatA));
	checkCuda(hipFree(d_MatB));
	checkCuda(hipFree(d_MatC));

	// free host memory
	free(h_A);
	free(h_B);
	free(hostRef);
	free(gpuRef);

	// reset device
	checkCuda(hipDeviceReset());

	return 0;
}