#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <>
#include <hip/hip_runtime.h>


#define N 300000
#define NSTREAM 4

// function for checking the CUDA runtime API results.
inline
void checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess)
	{
		printf_s("Error: %s : %d", __FILE__, __LINE__);
		printf_s("CUDA Runtime Error: %d: %s\n", result, hipGetErrorString(result));
		exit(1);
	}
#endif
}


__global__ void kernel_1()
{
	double sum = 0.0;

	for (int i = 0; i < N; i++)
	{
		sum = sum + tan(0.1) * tan(0.1);
	}
}

__global__ void kernel_2()
{
	double sum = 0.0;

	for (int i = 0; i < N; i++)
	{
		sum = sum + tan(0.1) * tan(0.1);
	}
}

__global__ void kernel_3()
{
	double sum = 0.0;

	for (int i = 0; i < N; i++)
	{
		sum = sum + tan(0.1) * tan(0.1);
	}
}

__global__ void kernel_4()
{
	double sum = 0.0;

	for (int i = 0; i < N; i++)
	{
		sum = sum + tan(0.1) * tan(0.1);
	}
}

int main(int argc, char **argv)
{
	int n_streams = NSTREAM;
	int isize = 1;
	int iblock = 1;
	int bigcase = 0;

	// get argument from command line
	if (argc > 1) n_streams = atoi(argv[1]);

	if (argc > 2) bigcase = atoi(argv[2]);

	float elapsed_time;

	int dev = 0;
	hipDeviceProp_t deviceProp;

	checkCuda(hipGetDeviceProperties(&deviceProp, dev));
	printf_s("> Using Device %d: %s with num_streams=%d\n", dev, deviceProp.name, n_streams);
	checkCuda(hipSetDevice(dev));

	// check if device support hyper-q
	if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5))
	{
		if (deviceProp.concurrentKernels == 0)
		{
			printf_s("> GPU does not support concurrent kernel execution (SM 3.5 or higher required)\n");
			printf_s("> CUDA kernel runs will be serialized\n");
		}
		else
		{
			printf_s("> GPU does not support HyperQ\n");
			printf_s("> CUDA kernel runs will have limited concurrency\n");
		}
	}

	printf_s("> Compute Capability %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

	// Allocate and initialize an array of stream handles
	hipStream_t *streams = (hipStream_t *)malloc(n_streams * sizeof(hipStream_t));

	for (int i = 0; i < n_streams; i++)
	{
		checkCuda(hipStreamCreate(&(streams[i])));
	}

	// run kernel with more threads
	if (bigcase == 1)
	{
		iblock = 512;
		isize = 1 << 18;
	}

	// set up execution configuration
	dim3 block(iblock);
	dim3 grid(isize / iblock);
	printf_s("> grid %d block %d\n", grid.x, block.x);

	// creat events
	hipEvent_t start, stop;
	checkCuda(hipEventCreate(&start));
	checkCuda(hipEventCreate(&stop));

	// record start event
	checkCuda(hipEventRecord(start, 0));

	// dispatch job with depth first ordering
	for (int i = 0; i < n_streams; i++)
	{
		kernel_1 <<<grid, block, 0, streams[i] >>>();
		kernel_2 <<<grid, block, 0, streams[i] >>>();
		kernel_3 <<<grid, block, 0, streams[i] >>>();
		kernel_4 <<<grid, block, 0, streams[i] >>>();
	}

	// record stop event
	checkCuda(hipEventRecord(stop, 0));
	checkCuda(hipEventSynchronize(stop));

	// calculate elapsed time
	checkCuda(hipEventElapsedTime(&elapsed_time, start, stop));
	printf_s("Measured time for parallel execution = %.3fs\n", elapsed_time / 1000.0f);

	// release all stream
	for (int i = 0; i < n_streams; i++)
	{
		checkCuda(hipStreamDestroy(streams[i]));
	}

	free(streams);

	// destroy events
	checkCuda(hipEventDestroy(start));
	checkCuda(hipEventDestroy(stop));

	// reset device
	checkCuda(hipDeviceReset());

	return EXIT_SUCCESS;
}