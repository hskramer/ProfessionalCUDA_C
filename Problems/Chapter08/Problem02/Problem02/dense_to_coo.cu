#include <stdio.h>
#include <stdlib.h>
#include <hipsparse.h>
#include <>
#include <hip/hip_runtime.h>
#include "common.h"

// function for checking the CUDA runtime API results.


void dense2coo(float *A, int M, int N, float **values, int **row_indices, int **col_indices)
{
	hipsparseHandle_t	A_handle = 0;
	hipsparseMatDescr_t	A_descr  = 0;


	float	 *d_A, *d_csrValA;
	int		 *d_nnzPerRow, *d_csrRowPtrA, *d_csrColIndA, *d_cooRowIndA;
	int		 nnzTotal;
	

	CHECK_CUSPARSE(hipsparseCreate(&A_handle));

	// construct a description of matrix A
	CHECK_CUSPARSE(hipsparseCreateMatDescr(&A_descr));
	CHECK_CUSPARSE(hipsparseSetMatType(A_descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
	CHECK_CUSPARSE(hipsparseSetMatIndexBase(A_descr, HIPSPARSE_INDEX_BASE_ZERO));

	CHECK(hipMalloc(&d_A, M * N * sizeof(float)));
	CHECK(hipMalloc(&d_nnzPerRow, M * sizeof(int)));

	CHECK_CUSPARSE(hipsparseSnnz(A_handle, HIPSPARSE_DIRECTION_ROW, M, N, A_descr, d_A, M, d_nnzPerRow, &nnzTotal));

	CHECK(hipMalloc(&d_csrRowPtrA,  (M + 1) * sizeof(int)));
	CHECK(hipMalloc(&d_csrValA,  nnzTotal   * sizeof(float)));
	CHECK(hipMalloc(&d_csrColIndA, nnzTotal * sizeof(float)));
	CHECK(hipMalloc(&d_cooRowIndA, nnzTotal * sizeof(float)));

	CHECK_CUSPARSE(hipsparseSdense2csr(A_handle, M, N, A_descr, d_A, M, d_nnzPerRow, d_csrValA, d_csrRowPtrA, d_csrColIndA));

	CHECK_CUSPARSE(hipsparseXcsr2coo(A_handle, d_csrRowPtrA, nnzTotal, M, d_cooRowIndA, HIPSPARSE_INDEX_BASE_ZERO));

	float *values = (float *)malloc(nnzTotal * sizeof(float));
	int   *row_i  = (int *)malloc(nnzTotal * sizeof(int));
	int   *col_i  = (int *)malloc(nnzTotal * sizeof(int));

	CHECK(hipMemcpy(values, d_csrValA, nnzTotal * sizeof(float), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(row_i, d_cooRowIndA, nnzTotal * sizeof(int), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(col_i, d_csrColIndA, nnzTotal * sizeof(int), hipMemcpyDeviceToHost));

}