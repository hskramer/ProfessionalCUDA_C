#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

// function for checking the CUDA runtime API results.
inline
void checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess)
	{
		printf_s("Error: %s : %d", __FILE__, __LINE__);
		printf_s("CUDA Runtime Error: %d: %s\n", result, hipGetErrorString(result));
		exit(1);
	}
#endif
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
	double  epsilon = 1.0E-8;
	bool bResult = true;
	for (int i = 0; i < N; i++)
	{
		if (hostRef[i] - gpuRef[i] > epsilon)
		{
			printf_s("Arrays do not match.\n");
			printf_s("host: %5.2f  gpu: %5.2f at current location %d\n", hostRef[i], gpuRef[i], i);
			bResult = false;
			break;
		}
	}

	if (bResult)
	{
		printf_s("Arrays match\n");
		return;
	}
}

void initialData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{
		ip[i] = (float)(rand() & 0xFF) / 100.0f;
	}

	return;
}


void sumArraysOnHost(float *A, float *B, float *C, const int n, int offset)
{
	for (int idx = offset, k = 0; idx < n; idx++, k++)
	{
		C[k] = A[idx] + B[idx];
	}
}

__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int k = i + offset;

	if (k < n) C[i] = A[k] + B[k];
}

__global__ void readOffset(float *A, float *B, float *C, const int n,
	int offset)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int k = i + offset;

	if (k < n) C[i] = A[k] + B[k];
}

int main(int argc, char **argv)
{
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;

	checkCuda(hipGetDeviceProperties(&deviceProp, dev));
	printf("%s starting reduction at ", argv[0]);
	printf("device %d: %s ", dev, deviceProp.name);
	checkCuda(hipSetDevice(dev));

	// set up array size
	int nElem = 1 << 22; // total number of elements to reduce
	printf(" with array size %d\n", nElem);
	size_t nBytes = nElem * sizeof(float);

	// set up offset for summary
	int blocksize = 512;
	int offset = 0;

	if (argc > 1) offset = atoi(argv[1]);

	if (argc > 2) blocksize = atoi(argv[2]);

	// execution configuration
	dim3 block(blocksize, 1);
	dim3 grid((nElem + block.x - 1) / block.x, 1);

	// allocate host memory
	float *h_a		= (float *)malloc(nBytes);
	float *h_b		= (float *)malloc(nBytes);
	float *hostRef	= (float *)malloc(nBytes);
	float *gpuRef	= (float *)malloc(nBytes);

	//  initialize host array
	initialData(h_a, nElem);
	memcpy(h_b, h_a, nBytes);

	//  summary at host side
	sumArraysOnHost(h_a, h_b, hostRef, nElem, offset);

	// allocate device memory
	float *d_a, *d_b, *d_c;
	checkCuda(hipMalloc((float**)&d_a, nBytes));
	checkCuda(hipMalloc((float**)&d_b, nBytes));
	checkCuda(hipMalloc((float**)&d_c, nBytes));

	// copy data from host to device
	checkCuda(hipMemcpy(d_a, h_a, nBytes, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(d_b, h_a, nBytes, hipMemcpyHostToDevice));

	//  kernel 1:
	warmup <<<grid, block >>>(d_a, d_b, d_c, nElem, offset);
	checkCuda(hipDeviceSynchronize());
	
	printf("warmup  <<< %4d, %4d >>> offset %4d\n", grid.x, block.x, offset);

	readOffset <<<grid, block >>>(d_a, d_b, d_c, nElem, offset);
	checkCuda(hipDeviceSynchronize());

	printf("readOffset  <<< %4d, %4d >>> offset %4d\n", grid.x, block.x, offset);

	// copy kernel result back to host side and check device results
	checkCuda(hipMemcpy(gpuRef, d_c, nBytes, hipMemcpyDeviceToHost));
	checkResult(hostRef, gpuRef, nElem - offset);

	// free host and device memory
	checkCuda(hipFree(d_a));
	checkCuda(hipFree(d_b));
	checkCuda(hipFree(d_c));
	free(h_a);
	free(h_b);

	// reset device
	checkCuda(hipDeviceReset());
	return EXIT_SUCCESS;
}