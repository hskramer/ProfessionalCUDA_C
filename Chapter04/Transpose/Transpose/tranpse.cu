#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <Windows.h>
#include <hip/hip_runtime.h>

/*
* Various memory access pattern optimizations applied to a matrix transpose
* kernel.
*/

inline
void checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess)
	{
		printf_s("Error: %s : %d", __FILE__, __LINE__);
		printf_s("CUDA Runtime Error: %d: %s\n", result, hipGetErrorString(result));
		exit(1);
	}
#endif
}


void initialData(float *in, const int size)
{
	for (int i = 0; i < size; i++)
	{
		in[i] = (float)(rand() & 0xFF) / 10.0f; //100.0f;
	}

	return;
}

void printData(float *in, const int size)
{
	for (int i = 0; i < size; i++)
	{
		printf_s("%dth element: %f\n", i, in[i]);
	}

	return;
}

void checkResult(float *hostRef, float *gpuRef, const int size)
{
	double epsilon = 1.0E-8;
	bool match = 1;

	for (int i = 0; i < size; i++)
	{
		if (abs(hostRef[i] - gpuRef[i]) > epsilon)
		{
			match = 0;
			printf_s("different on %dth element: host %f gpu %f\n", i, hostRef[i], gpuRef[i]);
			break;
		}

	}

	if (!match)  printf_s("Arrays do not match.\n\n");
}

void transposeHost(float *out, float *in, const int nx, const int ny)
{
	for (int iy = 0; iy < ny; ++iy)
	{
		for (int ix = 0; ix < nx; ++ix)
		{
			out[ix * ny + iy] = in[iy * nx + ix];
		}
	}
}

__global__ void warmup(float *out, float *in, const int nx, const int ny)
{
	unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

	if (ix < nx && iy < ny)
	{
		out[iy * nx + ix] = in[iy * nx + ix];
	}
}

// case 0 copy kernel: access data in rows, simulates the same amount of memory  ops as transpose with coalesced access
__global__ void copyRow(float *out, float *in, const int nx, const int ny)
{
	unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

	if (ix < nx && iy < ny)
	{
		out[iy * nx + ix] = in[iy * nx + ix];
	}
}

// case 1 copy kernel: access data in columns, simulates the same amount of memory  ops as transpose with strided access
__global__ void copyCol(float *out, float *in, const int nx, const int ny)
{
	unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

	if (ix < nx && iy < ny)
	{
		out[ix * ny + iy] = in[ix * ny + iy];
	}
}

// case 2 transpose kernel: read in rows and write in columns
__global__ void transposeNaiveRow(float *out, float *in, const int nx, const int ny)
{
	unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

	if (ix < nx && iy < ny)
	{
		out[ix * ny + iy] = in[iy * nx + ix];
	}
}

// case 3 transpose kernel: read in columns and write in rows
__global__ void transposeNaiveCol(float *out, float *in, const int nx, const int ny)
{
	unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

	if (ix < nx && iy < ny)
	{
		out[iy * nx + ix] = in[ix * ny + iy];
	}
}

// case 4 transpose kernel: read in rows and write in columns + unroll 4 blocks
__global__ void transposeUnroll4Row(float *out, float *in, const int nx, const int ny)
{
	unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
	unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

	unsigned int ti = iy * nx + ix; // access in rows
	unsigned int to = ix * ny + iy; // access in columns

	if (ix + 3 * blockDim.x < nx && iy < ny)
	{
		out[to] = in[ti];
		out[to + ny * blockDim.x] = in[ti + blockDim.x];
		out[to + ny * 2 * blockDim.x] = in[ti + 2 * blockDim.x];
		out[to + ny * 3 * blockDim.x] = in[ti + 3 * blockDim.x];
	}
}

// case 5 transpose kernel: read in columns and write in rows + unroll 4 blocks
__global__ void transposeUnroll4Col(float *out, float *in, const int nx, const int ny)
{
	unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
	unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

	unsigned int ti = iy * nx + ix; // access in rows
	unsigned int to = ix * ny + iy; // access in columns

	if (ix + 3 * blockDim.x < nx && iy < ny)
	{
		out[ti] = in[to];
		out[ti + blockDim.x] = in[to + blockDim.x * ny];
		out[ti + 2 * blockDim.x] = in[to + 2 * blockDim.x * ny];
		out[ti + 3 * blockDim.x] = in[to + 3 * blockDim.x * ny];
	}
}

/*
* case 6 :  transpose kernel: read in rows and write in colunms + diagonal
* coordinate transform
*/
__global__ void transposeDiagonalRow(float *out, float *in, const int nx, const int ny)
{
	unsigned int blk_y = blockIdx.x;
	unsigned int blk_x = (blockIdx.x + blockIdx.y) % gridDim.x;

	unsigned int ix = blockDim.x * blk_x + threadIdx.x;
	unsigned int iy = blockDim.y * blk_y + threadIdx.y;

	if (ix < nx && iy < ny)
	{
		out[ix * ny + iy] = in[iy * nx + ix];
	}
}

/*
* case 7 :  transpose kernel: read in columns and write in row + diagonal
* coordinate transform.
*/
__global__ void transposeDiagonalCol(float *out, float *in, const int nx, const int ny)
{
	unsigned int blk_y = blockIdx.x;
	unsigned int blk_x = (blockIdx.x + blockIdx.y) % gridDim.x;

	unsigned int ix = blockDim.x * blk_x + threadIdx.x;
	unsigned int iy = blockDim.y * blk_y + threadIdx.y;

	if (ix < nx && iy < ny)
	{
		out[iy * nx + ix] = in[ix * ny + iy];
	}
}

// main functions
int main(int argc, char **argv)
{
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	checkCuda(hipGetDeviceProperties(&deviceProp, dev));
	printf_s("%s starting transpose at ", argv[0]);
	printf_s("device %d: %s ", dev, deviceProp.name);
	checkCuda(hipSetDevice(dev));

	// set up array size 2048
	int nx = 1 << 11;
	int ny = 1 << 11;

	// select a kernel and block size
	int iKernel = 0;
	int blockx = 16;
	int blocky = 16;

	if (argc > 1) iKernel = atoi(argv[1]);

	if (argc > 2) blockx = atoi(argv[2]);

	if (argc > 3) blocky = atoi(argv[3]);

	if (argc > 4) nx = atoi(argv[4]);

	if (argc > 5) ny = atoi(argv[5]);

	printf_s(" with matrix nx %d ny %d with kernel %d\n", nx, ny, iKernel);
	size_t nBytes = nx * ny * sizeof(float);

	// execution configuration
	dim3 block(blockx, blocky);
	dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

	// allocate host memory
	float *h_A		= (float *)malloc(nBytes);
	float *hostRef	= (float *)malloc(nBytes);
	float *gpuRef	= (float *)malloc(nBytes);

	// initialize host array
	initialData(h_A, nx * ny);

	// transpose at host side
	transposeHost(hostRef, h_A, nx, ny);

	// allocate device memory
	float *d_A, *d_C;
	checkCuda(hipMalloc((float**)&d_A, nBytes));
	checkCuda(hipMalloc((float**)&d_C, nBytes));

	// copy data from host to device
	checkCuda(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));

	// warmup to avoide startup overhead
	clock_t	iStart, iStop;
	iStart = clock();

	warmup << <grid, block >> >(d_C, d_A, nx, ny);
	checkCuda(hipDeviceSynchronize());
	iStop = clock();
	float iElaps = float(iStop - iStart) / CLOCKS_PER_SEC;
	printf_s("warmup         elapsed %f sec\n", iElaps);
	checkCuda(hipGetLastError());

	// kernel pointer and descriptor
	void(*kernel)(float *, float *, int, int);
	char *kernelName;

	// set up kernel
	switch (iKernel)
	{
	case 0:
		kernel = &copyRow;
		kernelName = "CopyRow       ";
		break;

	case 1:
		kernel = &copyCol;
		kernelName = "CopyCol       ";
		break;

	case 2:
		kernel = &transposeNaiveRow;
		kernelName = "NaiveRow      ";
		break;

	case 3:
		kernel = &transposeNaiveCol;
		kernelName = "NaiveCol      ";
		break;

	case 4:
		kernel = &transposeUnroll4Row;
		kernelName = "Unroll4Row    ";
		grid.x = (nx + block.x * 4 - 1) / (block.x * 4);
		break;

	case 5:
		kernel = &transposeUnroll4Col;
		kernelName = "Unroll4Col    ";
		grid.x = (nx + block.x * 4 - 1) / (block.x * 4);
		break;

	case 6:
		kernel = &transposeDiagonalRow;
		kernelName = "DiagonalRow   ";
		break;

	case 7:
		kernel = &transposeDiagonalCol;
		kernelName = "DiagonalCol   ";
		break;
	}

	// run kernel
	iStart = clock();
	kernel << <grid, block >> >(d_C, d_A, nx, ny);
	checkCuda(hipDeviceSynchronize());
	iStop = clock();
    iElaps = float(iStop - iStart) / CLOCKS_PER_SEC;
	printf_s("%s elapsed %f sec <<< grid (%d,%d) block (%d,%d)>>> \n", kernelName, iElaps, grid.x, grid.y, block.x, block.y);
	checkCuda(hipGetLastError());

	// check kernel results
	if (iKernel > 1)
	{
		checkCuda(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
		checkResult(hostRef, gpuRef, nx * ny);
	}

	// free host and device memory
	checkCuda(hipFree(d_A));
	checkCuda(hipFree(d_C));
	free(h_A);
	free(hostRef);
	free(gpuRef);

	// reset device
	checkCuda(hipDeviceReset());
	return EXIT_SUCCESS;
}
