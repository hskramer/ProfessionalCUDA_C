#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

inline
void checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess)
	{
		printf("Error: %s : %d", __FILE__, __LINE__);
		printf("CUDA Runtime Error: %d: %s\n", result, hipGetErrorString(result));
		exit(1);
	}
#endif
}

__device__ float devData;

__global__ void checkGlobalVar()
{
	// display original value
	printf("Device global var: %f\n", devData);

	// alter the value
	devData += 2.0f;
}

int main(void)
{
	// initialize global variable
	float  value = 3.14f;

	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float)));

	printf_s("Host:   copied %f to the global variable\n", value);

	// invoke the kernel
	checkGlobalVar <<<1, 1 >>>();

	// copy the global variable back to the host
	checkCuda(hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float)));
	printf_s("Host:   the value changed by the kernel to %f\n", value);

	checkCuda(hipDeviceReset());

	return 0;
}