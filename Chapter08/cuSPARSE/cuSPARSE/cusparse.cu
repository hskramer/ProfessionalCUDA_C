#include <stdio.h>
#include <stdlib.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/*
* This is an example demonstrating usage of the cuSPARSE library to perform a
* sparse matrix-vector multiplication on randomly generated data.
*/

/*
* M = # of rows
* N = # of columns
*/
int M = 5120;
int N = 5120;

// function for checking the CUDA runtime API results.
inline
void checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess)
	{
		printf_s("Error: %s : %d", __FILE__, __LINE__);
		printf_s("CUDA Runtime Error: %d: %s\n", result, hipGetErrorString(result));
		exit(1);
	}
#endif
}


/*
* Generate a vector of length N with random single-precision floating-point
* values between 0 and 100.
*/
void generate_random_vector(int N, float **outX)
{
	int i;
	double rMax = (double)RAND_MAX;
	float *X = (float *)malloc(sizeof(float) * N);

	for (i = 0; i < N; i++)
	{
		int r = rand();
		double dr = (double)r;
		X[i] = (dr / rMax) * 100.0;
	}

	*outX = X;
}

/*
* Generate random dense matrix A in column-major order, while rounding some
* elements down to zero to ensure it is sparse.
*/
int generate_random_dense_matrix(int M, int N, float **outA)
{
	int i, j;
	double rMax = (double)RAND_MAX;
	float *A = (float *)malloc(sizeof(float) * M * N);
	int totalNnz = 0;

	for (j = 0; j < N; j++)
	{
		for (i = 0; i < M; i++)
		{
			int r = rand();
			float *curr = A + (j * M + i);

			if (r % 3 > 0)
			{
				*curr = 0.0f;
			}
			else
			{
				double dr = (double)r;
				*curr = (dr / rMax) * 100.0;
			}

			if (*curr != 0.0f)
			{
				totalNnz++;
			}
		}
	}

	*outA = A;
	return totalNnz;
}

int main(int argc, char **argv)
{
	int row;
	float *A, *dA;
	int *dNnzPerRow;
	float *dCsrValA;
	int *dCsrRowPtrA;
	int *dCsrColIndA;
	int totalNnz;
	float alpha = 3.0f;
	float beta = 4.0f;
	float *dX, *X;
	float *dY, *Y;

	hipsparseHandle_t handle = 0;
	hipsparseMatDescr_t descr = 0;

	// Generate input
	srand(9384);
	int trueNnz = generate_random_dense_matrix(M, N, &A);
	generate_random_vector(N, &X);
	generate_random_vector(M, &Y);

	// Create the cuSPARSE handle
	hipsparseCreate(&handle);

	// Allocate device memory for vectors and the dense form of the matrix A
	checkCuda(hipMalloc((void **)&dX, sizeof(float) * N));
	checkCuda(hipMalloc((void **)&dY, sizeof(float) * M));
	checkCuda(hipMalloc((void **)&dA, sizeof(float) * M * N));
	checkCuda(hipMalloc((void **)&dNnzPerRow, sizeof(int) * M));

	// Construct a descriptor of the matrix A by default it creates a general matrix with index base zero
	hipsparseCreateMatDescr(&descr);
	

	// Transfer the input vectors and dense matrix A to the device
	checkCuda(hipMemcpy(dX, X, sizeof(float) * N, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(dY, Y, sizeof(float) * M, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(dA, A, sizeof(float) * M * N, hipMemcpyHostToDevice));

	// Compute the number of non-zero elements in A
	hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, M, N, descr, dA, M, dNnzPerRow, &totalNnz);

	if (totalNnz != trueNnz)
	{
		fprintf_s(stderr, "Difference detected between cuSPARSE NNZ and true value: expected %d but got %d\n", trueNnz, totalNnz);
		return 1;
	}

	// Allocate device memory to store the sparse CSR representation of A
	checkCuda(hipMalloc((void **)&dCsrValA, sizeof(float) * totalNnz));
	checkCuda(hipMalloc((void **)&dCsrRowPtrA, sizeof(int) * (M + 1)));
	checkCuda(hipMalloc((void **)&dCsrColIndA, sizeof(int) * totalNnz));

	// Convert A from a dense formatting to a CSR formatting, using the GPU this takes the most time
	hipsparseSdense2csr(handle, M, N, descr, dA, M, dNnzPerRow, dCsrValA, dCsrRowPtrA, dCsrColIndA);

	// Perform matrix-vector multiplication with the CSR-formatted matrix A
	hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, M, N, totalNnz, &alpha, descr, dCsrValA, dCsrRowPtrA, dCsrColIndA, dX, &beta, dY);

	// Copy the result vector back to the host
	checkCuda(hipMemcpy(Y, dY, sizeof(float) * M, hipMemcpyDeviceToHost));

	for (row = 0; row < 10; row++)
	{
		printf_s("%2.2f\n", Y[row]);
	}

	printf_s("...\n");

	free(A);
	free(X);
	free(Y);

	checkCuda(hipFree(dX));
	checkCuda(hipFree(dY));
	checkCuda(hipFree(dA));
	checkCuda(hipFree(dNnzPerRow));
	checkCuda(hipFree(dCsrValA));
	checkCuda(hipFree(dCsrRowPtrA));
	checkCuda(hipFree(dCsrColIndA));

	hipsparseDestroyMatDescr(descr);
	hipsparseDestroy(handle);


	return 0;
}
