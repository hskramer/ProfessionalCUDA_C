#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


// function for checking the CUDA runtime API results.
inline
void checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess)
	{
		printf("Error: %s : %d, ", __FILE__, __LINE__);
		printf("CUDA Runtime Error: %d: %s\n", result, hipGetErrorString(result));
		exit(1);
	}
#endif
}


void initialData(float *ip, const int size)
{
	int i;

	for (i = 0; i < size; i++)
	{
		ip[i] = (float)(rand() & 0xFF) / 10.0f;
	}

	return;
}

void sumMatrixOnHost(float *A, float *B, float *C, const int nx,
	const int ny)
{
	float *ia = A;
	float *ib = B;
	float *ic = C;

	for (int iy = 0; iy < ny; iy++)
	{
		for (int ix = 0; ix < nx; ix++)
		{
			ic[ix] = ia[ix] + ib[ix];

		}

		ia += nx;
		ib += nx;
		ic += nx;
	}

	return;
}


void checkResult(float *hostRef, float *gpuRef, const int N)
{
	double epsilon = 1.0E-8;
	bool match = 1;

	for (int i = 0; i < N; i++)
	{
		if (abs(hostRef[i] - gpuRef[i]) > epsilon)
		{
			match = 0;
			printf("host %f gpu %f\n", hostRef[i], gpuRef[i]);
			break;
		}
	}

	if (match)
		printf("Arrays match.\n\n");
	else
		printf("Arrays do not match.\n\n");
}

// grid 1D block 1D
__global__ void sumMatrixOnGPU1D(float *MatA, float *MatB, float *MatC, int nx,	int ny)
{
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;

	if (ix < nx)
		for (int iy = 0; iy < ny; iy++)
		{
			int idx = iy * nx + ix;
			MatC[idx] = MatA[idx] + MatB[idx];
		}


}

int main(int argc, char **argv)
{
	printf("%s Starting...\n", argv[0]);

	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	checkCuda(hipGetDeviceProperties(&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	checkCuda(hipSetDevice(dev));

	// set up data size of matrix
	int nx = 1 << 14;
	int ny = 1 << 14;

	int nxy = nx * ny;
	int nBytes = nxy * sizeof(float);
	printf("Matrix size: nx %d ny %d\n", nx, ny);

	// malloc host memory
	float *h_A, *h_B, *hostRef, *gpuRef;
	h_A = (float *)malloc(nBytes);
	h_B = (float *)malloc(nBytes);
	hostRef = (float *)malloc(nBytes);
	gpuRef = (float *)malloc(nBytes);

	// initialize data at host side
	
	initialData(h_A, nxy);
	initialData(h_B, nxy);

	// add matrix at host side for result check
	sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);

	// malloc device global memory
	float *d_MatA, *d_MatB, *d_MatC;
	checkCuda(hipMalloc((void **)&d_MatA, nBytes));
	checkCuda(hipMalloc((void **)&d_MatB, nBytes));
	checkCuda(hipMalloc((void **)&d_MatC, nBytes));

	// transfer data from host to device
	checkCuda(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int dimx = 256;

	dim3 block(dimx, 1);
	dim3 grid((nx + block.x - 1) / block.x, 1);

	sumMatrixOnGPU1D << <grid, block >> >(d_MatA, d_MatB, d_MatC, nx, ny);
	checkCuda(hipDeviceSynchronize());

	printf("sumMatrixOnGPU1D <<<(%d,%d), (%d,%d)>>>\n", grid.x, grid.y, block.x, block.y);

	// Check kernel error
	checkCuda(hipGetLastError());

	// copy kernel result back to host side
	checkCuda(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost));

	// check device results
	checkResult(hostRef, gpuRef, nxy);

	// free device global memory
	checkCuda(hipFree(d_MatA));
	checkCuda(hipFree(d_MatB));
	checkCuda(hipFree(d_MatC));

	// free host memory
	free(h_A);
	free(h_B);
	free(hostRef);
	free(gpuRef);

	// reset device
	checkCuda(hipDeviceReset());

	return 0;
}
