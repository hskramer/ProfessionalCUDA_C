#include "hip/hip_runtime.h"
#include "common.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

float M_PI = 3.141592653589f;

int nprints = 30;

/*
* Create N fake samplings along the function cos(x). These samplings will be
* stored as single-precision floating-point values.
*/
void generate_fake_samples(int N, float **out)
{
	int i;
	float *result = (float *)malloc(sizeof(float) * N);
	double delta = M_PI / 20.0;

	for (i = 0; i < N; i++)
	{
		result[i] = cos(i * delta);
	}

	*out = result;
}

/*
* Convert a real-valued vector r of length Nto a complex-valued vector.
*/
void real_to_complex(float *r, hipfftComplex **complx, int N)
{
	int i;
	(*complx) = (hipfftComplex *)malloc(sizeof(hipfftComplex) * N);
		
	for (i = 0; i < N; i++)
	{
		(*complx)[i].x = r[i];
		(*complx)[i].y = 0;
	}
}

int main(int argc, char **argv)
{
	int		i;
	int		N = 2048;
	float	*samples;

	hipfftHandle		plan = 0;
	hipfftComplex	*d_ComplexSamples, *complexSamples, *complexFreq;


	// generate are data for the FFT
	generate_fake_samples(N, &samples);
	real_to_complex(samples, &complexSamples, N);

	complexFreq = (hipfftComplex *)malloc(N * sizeof(hipfftComplex));
	if (!complexFreq)
	{
		printf_s("host memory allocation failed");
		return EXIT_FAILURE;
	}

	printf_s("Initial Samples:\n");

	for (i = 0; i < nprints; i++)
	{
		printf_s("  %2.4f\n", samples[i]);
	}

	printf_s("  ...\n");

	// setup the cuFFT plan
	CHECK_CUFFT(hipfftPlan1d(&plan, N, HIPFFT_C2C, 1));

	// allocate device memory and transfer to device
	CHECK(hipMalloc((void **)&d_ComplexSamples, N * sizeof(hipfftComplex)));
	CHECK(hipMemcpy(d_ComplexSamples, complexSamples, N * sizeof(hipfftComplex), hipMemcpyHostToDevice));

	// execute are FFT forward then immediately reverse it
	CHECK_CUFFT(hipfftExecC2C(plan, d_ComplexSamples, d_ComplexSamples , HIPFFT_FORWARD));

	CHECK_CUFFT(hipfftExecC2C(plan, d_ComplexSamples, d_ComplexSamples, HIPFFT_BACKWARD));

	// retrieve the results from the gpu
	CHECK(hipMemcpy(complexFreq, d_ComplexSamples, N * sizeof(hipfftComplex), hipMemcpyDeviceToHost));

	printf_s("Fourier Coefficients:\n");

	float maximum = 0.0f;
	for (i = 0; i < N; i++)
	{
		if (fabs(complexFreq[i].x) > maximum)
		{
			maximum = fabs(complexFreq[i].x);
		}
	}


	for (i = 0; i < nprints; i++)
	{
		printf_s("  %d: (%2.4f, %2.4f)\n", i + 1, complexFreq[i].x / maximum, complexFreq[i].y / maximum);
	}

	printf_s("  ...\n");

	free(complexSamples);
	free(complexFreq);
	free(samples);

	CHECK(hipFree(d_ComplexSamples));
	CHECK_CUFFT(hipfftDestroy(plan));

	return EXIT_SUCCESS;

}