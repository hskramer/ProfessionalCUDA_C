#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


/* The CUDA Programming guide makes it clear that read/write on words of 1,2,4,8, or 16 bytes will compile to a
* single memory instruction this was clear when using nsight and nvprof which showed 100% load/store efficiency
* compared to 50% for the non-aligned version.
*/

#define LEN 1<<24

struct __align__(8) innerStruct
{
	float x;
	float y;
};

inline
void checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess)
	{
		printf_s("Error: %s : %d", __FILE__, __LINE__);
		printf_s("CUDA Runtime Error: %d: %s\n", result, hipGetErrorString(result));
		exit(1);
	}
#endif
}


void initialInnerStruct(innerStruct *ip, int size)
{
	for (int i = 0; i < size; i++)
	{
		ip[i].x = (float)(rand() & 0xFF) / 100.0f;
		ip[i].y = (float)(rand() & 0xFF) / 100.0f;
	}

	return;
}

void testInnerStructHost(innerStruct *A, innerStruct *C, const int N)
{
	for (int idx = 0; idx < N; idx++)
	{
		C[idx].x = A[idx].x + 10.f;
		C[idx].y = A[idx].y + 20.f;
	}

	return;
}

void checkInnerStruct(innerStruct *hostRef, innerStruct *gpuRef, const int N)
{
	double epsilon = 1.0E-8;
	bool match = 1;

	for (int i = 0; i < N; i++)
	{
		if (abs(hostRef[i].x - gpuRef[i].x) > epsilon)
		{
			match = 0;
			printf_s("different on %dth element: host %f gpu %f\n", i, hostRef[i].x, gpuRef[i].x);
			break;
		}

		if (abs(hostRef[i].y - gpuRef[i].y) > epsilon)
		{
			match = 0;
			printf_s("different on %dth element: host %f gpu %f\n", i, hostRef[i].y, gpuRef[i].y);
			break;
		}
	}

	if (!match)  printf("Arrays do not match.\n\n");
}

__global__ void testInnerStruct(innerStruct *data, innerStruct * result, const int N)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		innerStruct tmp = data[i];
		tmp.x += 10.f;
		tmp.y += 20.f;
		result[i] = tmp;
	}
}

__global__ void warmup(innerStruct *data, innerStruct * result, const int N)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		innerStruct tmp = data[i];
		tmp.x += 10.f;
		tmp.y += 20.f;
		result[i] = tmp;
	}
}

int main(int argc, char **argv)
{
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;

	checkCuda(hipGetDeviceProperties(&deviceProp, dev));
	printf_s("%s test struct of array at ", argv[0]);
	printf_s("device %d: %s \n", dev, deviceProp.name);

	checkCuda(hipSetDevice(dev));

	// allocate host memory
	int nElem = LEN;
	size_t nBytes = nElem * sizeof(innerStruct);

	innerStruct     *h_A = (innerStruct *)malloc(nBytes);
	innerStruct *hostRef = (innerStruct *)malloc(nBytes);
	innerStruct *gpuRef = (innerStruct *)malloc(nBytes);

	// initialize host array
	initialInnerStruct(h_A, nElem);
	testInnerStructHost(h_A, hostRef, nElem);

	// allocate device memory
	innerStruct *d_A, *d_C;
	checkCuda(hipMalloc((innerStruct**)&d_A, nBytes));
	checkCuda(hipMalloc((innerStruct**)&d_C, nBytes));

	// copy data from host to device
	checkCuda(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));

	// set up offset for summaryAU: It is blocksize not offset.
	int blocksize = 256;

	if (argc > 1) blocksize = atoi(argv[1]);

	// execution configuration
	dim3 block(blocksize, 1);
	dim3 grid((nElem + block.x - 1) / block.x, 1);

	// kernel 1: warmup

	warmup << <grid, block >> >(d_A, d_C, nElem);
	checkCuda(hipDeviceSynchronize());

	printf_s("warmup      <<< %3d, %3d >>>\n", grid.x, block.x);
	checkCuda(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
	checkInnerStruct(hostRef, gpuRef, nElem);

	// kernel 2: testInnerStruct

	testInnerStruct << <grid, block >> >(d_A, d_C, nElem);
	checkCuda(hipDeviceSynchronize());

	printf_s("innerstruct <<< %3d, %3d >>>\n", grid.x, block.x);
	checkCuda(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
	checkInnerStruct(hostRef, gpuRef, nElem);

	// free memories both host and device
	checkCuda(hipFree(d_A));
	checkCuda(hipFree(d_C));
	free(h_A);
	free(hostRef);
	free(gpuRef);

	// reset device
	checkCuda(hipDeviceReset());

	return EXIT_SUCCESS;
}
