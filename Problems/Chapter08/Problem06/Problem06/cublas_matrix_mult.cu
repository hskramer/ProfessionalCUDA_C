#include "common.h"
#include <stdio.h>
#include <stdlib.h>
#include <>
#include <hip/hip_runtime.h>
#include <hipblas.h>

int M = 1048;
int N = 1048;


void generate_random_dense_matrix(int M, int N, float **outA)
{
	int i, j;
	double rMax = (double)RAND_MAX;
	float *A = (float *)malloc(sizeof(float) * M * N);

	// For each column
	for (j = 0; j < N; j++)
	{
		// For each row
		for (i = 0; i < M; i++)
		{
			double dr = (double)rand();
			A[j * M + i] = (dr / rMax) * 100.0;
		}
	}

	*outA = A;
}

int main(int argc, char **argv)
{
	float	*A, *d_A;
	float	*B, *d_B;
	float	*C, *d_C;
	float	alpha = 3.0f, beta = 5.0;
	hipblasHandle_t	handle =0;

	// obtain out two matrices
	srand(2468);
	generate_random_dense_matrix(M, N, &A);
	generate_random_dense_matrix(M, N, &B);
	
	C = (float *)malloc(sizeof(float) * M * N);

	// allocate device memory
	CHECK(hipMalloc((void **)&d_A, M * N * sizeof(float)));
	CHECK(hipMalloc((void **)&d_B, M * N * sizeof(float)));
	CHECK(hipMalloc((void **)&d_C, M * N * sizeof(float)));

	// Create the handle and setup the matrices
	CHECK_CUBLAS(hipblasCreate(&handle));
	CHECK_CUBLAS(hipblasSetMatrix(M, N, sizeof(float), A, M, d_A, M));
	CHECK_CUBLAS(hipblasSetMatrix(M, N, sizeof(float), B, M, d_B, M));
	CHECK_CUBLAS(hipblasSetMatrix(M, N, sizeof(float), C, M, d_C, M));

	// Use cuBLAS for my matrix multiplication
	CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, N, &alpha, d_A, M, d_B, M, &beta, d_C, M));

	// Retrieve the results of the multiplication
	CHECK_CUBLAS(hipblasGetMatrix(M, N, sizeof(float), d_C, M, C, M));

	int i, j;

	for (j = 0; j < 10; j++)
	{
		for (i = 0; i < 10; i++)
		{
			printf_s("%2.2f ", C[j * M + i]);
		}
		printf_s("...\n");
	}

	printf_s("....\n");

	free(A);
	free(B);
	free(C);

	CHECK(hipFree(d_A));
	CHECK(hipFree(d_B));
	CHECK(hipFree(d_C));

	CHECK_CUBLAS(hipblasDestroy(handle));

	return EXIT_SUCCESS;

}