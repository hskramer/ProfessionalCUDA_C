#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <>
#include <hip/hip_runtime.h>


#define DIM		 128
#define SMEMDIM	 4

// function for checking the CUDA runtime API results.
inline
void checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess)
	{
		printf_s("Error: %s : %d", __FILE__, __LINE__);
		printf_s("CUDA Runtime Error: %d: %s\n", result, hipGetErrorString(result));
		exit(1);
	}
#endif
}

// Recursive Implementation of Interleaved Pair Approach
int recursiveReduce(int *data, int const size)
{
	if (size == 1) return data[0];

	int const stride = size / 2;

	for (int i = 0; i < stride; i++)
		data[i] += data[i + stride];

	return recursiveReduce(data, stride);
}

__inline__ __device__ int warpReduce(int localSum)
{
	localSum += __shfl_down(localSum, 16);
	localSum += __shfl_down(localSum, 8);
	localSum += __shfl_down(localSum, 4);
	localSum += __shfl_down(localSum, 2);
	localSum += __shfl_down(localSum, 1);

	return localSum;
}

__inline__ __device__ int warpReduceFor(int localSum)
{
	for (int i = 16; i > 0; i >>= 1)
		localSum += __shfl_down(localSum, i);

	return localSum;
}

__global__ void reduceShfl(int *g_idata, int *g_odata, unsigned int n)
{
	// shared memory for each warp sum
	__shared__ int smem[SMEMDIM];

	// boundary check
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= n) return;

	// calculate lane index and warp index
	int laneIdx = threadIdx.x % warpSize;
	int warpIdx = threadIdx.x / warpSize;

	// blcok-wide warp reduce
	int localSum = warpReduceFor(g_idata[idx]); // just testing warpReduceFor

	// save warp sum to shared memory
	if (laneIdx == 0) smem[warpIdx] = localSum;

	// block synchronization
	__syncthreads();

	// last warp reduce
	if (threadIdx.x < warpSize) localSum = (threadIdx.x < SMEMDIM) ? smem[laneIdx] : 0;

	if (warpIdx == 0) localSum = warpReduce(localSum);

	// write result for this block to global mem
	if (threadIdx.x == 0) g_odata[blockIdx.x] = localSum;
}


int main(int argc, char **argv)
{
	// set up device
	int  dev = 0;
	hipDeviceProp_t  deviceProp;

	checkCuda(hipGetDeviceProperties(&deviceProp, dev));
	printf_s("%s starting reduction on ", argv[0]);
	printf_s("device %d: %s ", dev, deviceProp.name);
	checkCuda(hipSetDevice(dev));

	bool bResult = false;

	// initialization
	int lShft = 20;

	if (argc > 1) lShft = atoi(argv[1]);

	int nElem = 1 << lShft;
	printf_s("    with array size %d  ", nElem);

	// execution configuration
	int  blocksize = DIM;

	dim3  block(blocksize, 1);
	dim3  grid((nElem + block.x - 1) / block.x, 1);
	printf_s("grid %d block %d\n", grid.x, block.x);

	// allocate host memory
	size_t  nBytes = nElem * sizeof(int);

	int  *h_idata = (int *)malloc(nBytes);
	int  *h_odata = (int *)malloc(grid.x * sizeof(int));
	int  *tmp	  = (int *)malloc(nBytes);

	// initialize the array
	for (int i = 0; i < nElem; i++)
		h_idata[i] = (int)(rand() & 0xFF);

	memcpy(tmp, h_idata, nBytes);

	// allocate device memory
	int  *d_idata, *d_odata;
	checkCuda(hipMalloc(&d_idata, nBytes));
	checkCuda(hipMalloc(&d_odata, grid.x * sizeof(int)));

	// cpu reduction
	int cpu_sum = recursiveReduce(tmp, nElem);
	printf_s("cpu reduce          : %d\n", cpu_sum);

	
	// reduce Shfl
	checkCuda(hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice));
	reduceShfl <<<grid.x, block >>>(d_idata, d_odata, nElem);
	checkCuda(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost));

	int gpu_sum = 0;
	for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

	bResult = (gpu_sum == cpu_sum);

	if (!bResult) printf_s("Test failed!\n");

	printf_s("reduceShfl          : %d <<<grid %d block %d>>>\n", gpu_sum, grid.x, block.x);

	// free host memory
	free(h_idata);
	free(h_odata);
	free(tmp);

	// free device memory
	checkCuda(hipFree(d_idata));
	checkCuda(hipFree(d_odata));

	// reset device
	checkCuda(hipDeviceReset());

	return EXIT_SUCCESS;

}