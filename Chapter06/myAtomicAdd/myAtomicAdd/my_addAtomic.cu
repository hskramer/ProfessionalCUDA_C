#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>


// function for checking the CUDA runtime API results.
inline
void checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess)
	{
		printf_s("Error: %s : %d", __FILE__, __LINE__);
		printf_s("CUDA Runtime Error: %d: %s\n", result, hipGetErrorString(result));
		exit(1);
	}
#endif
}



__device__ int myAtomicAdd(int *addr, int incr)
{
	// Create an initial guess for the value stored at *address.
	int guess = *addr;
	int oldValue = atomicCAS(addr, guess, guess + incr);

	// Loop while the guess is incorrect.
	while (oldValue != guess)
	{
		guess = oldValue;
		oldValue = atomicCAS(addr, guess, guess + incr);
	}

	return oldValue;
}

__global__ void kernel(int *sharedInteger)
{
	myAtomicAdd(sharedInteger, 1);
}

int main(int argc, char **argv)
{
	int  h_sharedInt;
	int  *d_sharedInt;

	checkCuda(hipMalloc(&d_sharedInt, sizeof(int)));
	checkCuda(hipMemset(d_sharedInt, 0x00, sizeof(int)));

	kernel <<<4, 128 >>> (d_sharedInt);

	checkCuda(hipMemcpy(&h_sharedInt, d_sharedInt, sizeof(int), hipMemcpyDeviceToHost));
	printf_s("4 x 128 increments led to value of %d\n", h_sharedInt);

	return 0;

}